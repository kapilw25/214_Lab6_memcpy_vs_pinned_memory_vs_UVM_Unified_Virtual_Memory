#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int numElements = 50000; // Size of the vectors
    size_t size = numElements * sizeof(float);
    float *h_A, *h_B, *h_C; // Host vectors
    float *d_A, *d_B, *d_C; // Device vectors

    // Allocate pinned host memory
    hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, size, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, size, hipHostMallocDefault);

    // Initialize host vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Start recording the execution time
    auto start = std::chrono::high_resolution_clock::now();

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Stop recording the execution time
    hipDeviceSynchronize();
    auto finish = std::chrono::high_resolution_clock::now();

    // Copy result vector from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate and print the execution time
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free pinned host memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
