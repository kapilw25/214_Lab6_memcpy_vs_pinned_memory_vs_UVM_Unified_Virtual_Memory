#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int numElements = 50000; // Size of the vectors
    size_t size = numElements * sizeof(float);
    float *h_A, *h_B, *h_C; // Unified Memory pointers

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&h_A, size);
    hipMallocManaged(&h_B, size);
    hipMallocManaged(&h_C, size);

    // Optional: Set Memory Advise Hints
    int device = -1;
    hipGetDevice(&device);
    hipMemAdvise(h_A, size, hipMemAdviseSetPreferredLocation, device);
    hipMemAdvise(h_B, size, hipMemAdviseSetPreferredLocation, device);
    hipMemAdvise(h_C, size, hipMemAdviseSetPreferredLocation, device);

    // Initialize vectors in Unified Memory
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Start recording the execution time
    auto start = std::chrono::high_resolution_clock::now();

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(h_A, h_B, h_C, numElements);

    // Stop recording the execution time
    hipDeviceSynchronize();
    auto finish = std::chrono::high_resolution_clock::now();

    // Calculate and print the execution time
    std::chrono::duration<double> elapsed = finish - start;
    std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

    // Free Unified Memory
    hipFree(h_A);
    hipFree(h_B);
    hipFree(h_C);

    return 0;
}
